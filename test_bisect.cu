/*
 * Copyright (C) Marlin.2024 Elias Frantar (elias.frantar@ist.ac.at)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <iostream>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <vector>
#include <cmath>

__host__ __device__ inline int ceildiv(int a, int b) {
  return (a + b - 1) / b;
}

template <typename T, int n>
struct Vec {
  T elems[n];
  __device__ T& operator[](int i) {
    return elems[i];
  }
};

using I4 = Vec<int, 4>;
using FragA = Vec<half2, 4>;
using FragB = Vec<half2, 2>;
using FragC = Vec<float, 4>;
using FragS = Vec<half2, 1>;

__device__ inline void cp_async4_pred(void* smem_ptr, const void* glob_ptr, bool pred = true) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "{\n"
    "   .reg .pred p;\n"
    "   setp.ne.b32 p, %0, 0;\n"
    "   @p cp.async.cg.shared.global [%1], [%2], %3;\n"
    "}\n" :: "r"((int) pred), "r"(smem), "l"(glob_ptr), "n"(BYTES)
  );
}

__device__ inline void cp_async4_stream(void* smem_ptr, const void* glob_ptr) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "{\n"
    "   .reg .b64 p;\n"
    "   createpolicy.fractional.L2::evict_first.b64 p, 1.0;"
    "   cp.async.cg.shared.global.L2::cache_hint [%0], [%1], %2, p;\n"
    "}\n" :: "r"(smem), "l"(glob_ptr), "n"(BYTES)
  );
}

__device__ inline void cp_async_fence() {
  asm volatile("cp.async.commit_group;\n" ::);
}

template <int n>
__device__ inline void cp_async_wait() {
  asm volatile("cp.async.wait_group %0;\n" :: "n"(n));
}

__device__ inline void mma(const FragA& a_frag, const FragB& frag_b, FragC& frag_c) {
  const uint32_t* a = reinterpret_cast<const uint32_t*>(&a_frag);
  const uint32_t* b = reinterpret_cast<const uint32_t*>(&frag_b);
  float* c = reinterpret_cast<float*>(&frag_c);
  asm volatile(
    "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
    "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
    : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
    :  "r"(a[0]),  "r"(a[1]),  "r"(a[2]),  "r"(a[3]),  "r"(b[0]),  "r"(b[1]),
       "f"(c[0]),  "f"(c[1]),  "f"(c[2]),  "f"(c[3])
  );
}

__device__ inline void ldsm4(FragA& frag_a, const void* smem_ptr) {
  uint32_t* a = reinterpret_cast<uint32_t*>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0,%1,%2,%3}, [%4];\n"
    : "=r"(a[0]), "=r"(a[1]), "=r"(a[2]), "=r"(a[3]) : "r"(smem)
  );
}

template <int lut>
__device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile(
    "lop3.b32 %0, %1, %2, %3, %4;\n"
    : "=r"(res) : "r"(a), "r"(b), "r"(c), "n"(lut)
  );
  return res;
}

__device__ inline FragB dequant(int q) {
  const int LO = 0x000f000f;
  const int HI = 0x00f000f0;
  const int EX = 0x64006400;
  int lo = lop3<(0xf0 & 0xcc) | 0xaa>(q, LO, EX);
  int hi = lop3<(0xf0 & 0xcc) | 0xaa>(q, HI, EX);
  const int SUB = 0x64086408;
  const int MUL = 0x2c002c00;
  const int ADD = 0xd480d480;
  FragB frag_b;
  frag_b[0] = __hsub2(
    *reinterpret_cast<half2*>(&lo),
    *reinterpret_cast<const half2*>(&SUB)
  );
  frag_b[1] = __hfma2(
    *reinterpret_cast<half2*>(&hi),
    *reinterpret_cast<const half2*>(&MUL), *reinterpret_cast<const half2*>(&ADD)
  );
  return frag_b;
}

__device__ inline void scale(FragB& frag_b, FragS& frag_s, int i) {
  half2 s = __half2half2(reinterpret_cast<__half*>(&frag_s)[i]);
  frag_b[0] = __hmul2(frag_b[0], s);
  frag_b[1] = __hmul2(frag_b[1], s);
}

__device__ inline void barrier_acquire(int* lock, int count) {
  if (threadIdx.x == 0) {
    int state = -1;
    do
      asm volatile ("ld.global.acquire.gpu.b32 %0, [%1];\n" : "=r"(state) : "l"(lock));
    while (state != count);
  }
  __syncthreads();
}

__device__ inline void barrier_release(int* lock, bool reset = false) {
  __syncthreads();
  if (threadIdx.x == 0) {
    if (reset) {
      lock[0] = 0;
      return;
    }
    int val = 1;
    asm volatile ("fence.acq_rel.gpu;\n");
    asm volatile ("red.relaxed.gpu.global.add.s32 [%0], %1;\n" : : "l"(lock), "r"(val));
  }
}

template <
  const int threads,
  const int thread_m_blocks,
  const int thread_n_blocks,
  const int thread_k_blocks,
  const int stages,
  const int group_blocks = -1
>
__global__ void Marlin(
  const int4* __restrict__ A,
  const int4* __restrict__ B,
        int4* __restrict__ C,
  const int4* __restrict__ s,
  int  prob_m,
  int  prob_n,
  int  prob_k,
  int* locks
) {
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);
  if (group_blocks != -1)
    iters = (group_blocks / thread_k_blocks) * ceildiv(iters, (group_blocks / thread_k_blocks));

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters;
  int slice_count = 0;
  int slice_idx;

  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  auto init_slice = [&] () {
    slice_iters = iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel)
      slice_iters = 0;
    if (slice_iters == 0)
      return;
    if (slice_row + slice_iters > k_tiles)
      slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0)
        slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0)
          slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  int a_gl_stride = prob_k / 8;
  constexpr int a_sh_stride = 16 * thread_k_blocks / 8;
  constexpr int a_gl_rd_delta_o = 16 * thread_k_blocks / 8;
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o);
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o);
  constexpr int a_sh_rd_delta_o = 2 * ((threads / 32) / (thread_n_blocks / 4));
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16;
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks);
  constexpr int a_sh_wr_iters = (a_sh_stage + a_sh_wr_delta - 1) / a_sh_wr_delta;

  int b_gl_stride = 16 * prob_n / 32;
  constexpr int b_sh_stride = 32 * thread_n_blocks / 4;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride);
  constexpr int b_sh_wr_delta = threads;
  constexpr int b_sh_rd_delta = threads;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_sh_stage = s_sh_stride;
  int s_gl_rd_delta = s_gl_stride;

  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
  int a_sh_rd = a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 2 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x;
  int b_sh_rd = threadIdx.x;

  int s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) + s_sh_stride * slice_col + threadIdx.x;
  int s_sh_wr = threadIdx.x;
  int s_sh_rd;
  if (group_blocks != -1)
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + (threadIdx.x % 32) / 4;
  else
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + (threadIdx.x % 32) % 4;

  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  auto transform_a = [&] (int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
    #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] = transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_s = sh_b + (stages * b_sh_stage);
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];

  auto zero_accums = [&] () {
    #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float*>(frag_c)[i] = 0;
  };

  auto fetch_to_shared = [&] (int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
      #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
          &sh_a_stage[a_sh_wr_trans[i]],
          &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
          a_sh_wr_pred[i]
        );
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
      #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
        cp_async4_stream(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr], B_ptr[i]);
        B_ptr[i] += b_gl_rd_delta_o;
      }
      if (group_blocks != -1 && pipe % (group_blocks / thread_k_blocks) == 0) {
        int4* sh_s_stage = sh_s + s_sh_stage * pipe;
        if (s_sh_wr_pred)
          cp_async4_stream(&sh_s_stage[s_sh_wr], &s[s_gl_rd]);
        s_gl_rd += s_gl_rd_delta;
      }
    }
    cp_async_fence();
  };

  auto wait_for_stage = [&] () {
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  auto fetch_to_registers = [&] (int k, int pipe) {
    if (group_blocks != -1) {
      int4* sh_s_stage = sh_s + s_sh_stage * ((group_blocks / thread_k_blocks) * (pipe / (group_blocks / thread_k_blocks)));
      reinterpret_cast<int4*>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd];
    }
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
    #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4(frag_a[k % 2][i], &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4* sh_b_stage = sh_b + b_sh_stage * pipe;
    frag_b_quant[k % 2] = *reinterpret_cast<I4*>(&sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd]);
  };

  auto matmul = [&] (int k) {
    #pragma unroll
    for (int j = 0; j < 4; j++) {
      int b_quant = frag_b_quant[k % 2][j];
      int b_quant_shift = b_quant >> 8;
      FragB frag_b0 = dequant(b_quant);
      if (group_blocks != -1)
        scale(frag_b0, frag_s[k % 2][j], 0);
      FragB frag_b1 = dequant(b_quant_shift);
      if (group_blocks != -1)
        scale(frag_b1, frag_s[k % 2][j], 1);
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  auto thread_block_reduce = [&] () {
    constexpr int red_off = threads / b_sh_stride / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride;
      constexpr int red_sh_stride = b_sh_stride * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);

      #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
        #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
            #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr = red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float* c_rd = reinterpret_cast<float*>(&sh[red_sh_delta * j + red_sh_rd]);
                float* c_wr = reinterpret_cast<float*>(&sh[red_sh_wr]);
                #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] += c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] = reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
          #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float* c_rd = reinterpret_cast<float*>(&sh[red_sh_delta * i + red_sh_rd]);
            #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] += c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  auto global_reduce = [&] (bool first = false, bool last = false) {
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 4 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) + 4 * (threadIdx.x / 32) + threadIdx.x % 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      int c_sh_wr = threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
        #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(
            &sh[c_sh_wr + c_sh_wr_delta * i],
            &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)],
            i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m
          );
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

      #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
            #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<float*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] += __half2float(
                reinterpret_cast<__half*>(&c_red)[j]
              );
            }
          }
          if (!last) {
            int4 c;
            #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<__half*>(&c)[j] = __float2half(
                reinterpret_cast<float*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)]
              );
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] = c;
          }
        }
      }
    }
  };

  auto write_result = [&] () {
    int c_gl_stride = prob_n / 8;
    constexpr int c_sh_stride = 2 * thread_n_blocks + 1;
    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int c_sh_rd_delta = c_sh_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;
    int c_sh_wr = (4 * c_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    c_sh_wr += 32 * (threadIdx.x / 32);
    int c_sh_rd = c_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    auto write = [&] (int idx, float c0, float c1, FragS& s) {
      half2 res = __halves2half2(__float2half(c0), __float2half(c1));
      if (group_blocks == -1)
        res = __hmul2(res, s[0]);
      ((half2*) sh)[idx] = res;
    };
    if (threadIdx.x / 32 < thread_n_blocks / 4) {
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = c_sh_wr + 8 * j;
          write(wr + (4 * c_sh_stride) * 0 + 0, frag_c[i][j][0][0], frag_c[i][j][0][1], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 8 + 0, frag_c[i][j][0][2], frag_c[i][j][0][3], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 0 + 4, frag_c[i][j][1][0], frag_c[i][j][1][1], frag_s[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * c_sh_stride) * 8 + 4, frag_c[i][j][1][2], frag_c[i][j][1][3], frag_s[j / 2][2 * (j % 2) + 1]);
        }
        c_sh_wr += 16 * (4 * c_sh_stride);
      }
    }
    __syncthreads();

    #pragma unroll
    for (int i = 0; i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks)); i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  auto start_pipes = [&] () {
    #pragma unroll
    for (int i = 0; i < stages - 1; i++)
      fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();

  while (slice_iters) {
    #pragma unroll
    for (int pipe = 0; pipe < stages;) {
      #pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe, slice_iters >= stages);
          pipe++;
          wait_for_stage();
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0)
        break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;

    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      if (group_blocks == -1 && last) {
        if (s_sh_wr_pred)
          cp_async4_stream(&sh_s[s_sh_wr], &s[s_gl_rd]);
        cp_async_fence();
      }
      thread_block_reduce();
      if (group_blocks == -1 && last) {
        cp_async_wait<0>();
        __syncthreads();
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
          reinterpret_cast<int4*>(&frag_s)[0] = sh_s[s_sh_rd + 0];
          reinterpret_cast<int4*>(&frag_s)[1] = sh_s[s_sh_rd + 4];
        }
      }
      if (slice_count > 1) {
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last)
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
        #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
          #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++)
            B_ptr[i] -= b_gl_stride;
        }
        s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
        start_pipes();
      }
    }
  }
}

const int THREADS = 256;
const int STAGES = 4;
const int SHARED_MEM = 96 * 1024;

#define CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, GROUP_BLOCKS) \
  else if ( \
    thread_m_blocks == THREAD_M_BLOCKS && thread_n_blocks == THREAD_N_BLOCKS && thread_k_blocks == THREAD_K_BLOCKS && \
    group_blocks == GROUP_BLOCKS \
  ) { \
    hipFuncSetAttribute( \
      reinterpret_cast<const void*>(Marlin<THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>), \
      hipFuncAttributeMaxDynamicSharedMemorySize, \
      SHARED_MEM \
    ); \
    Marlin< \
      THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS \
    ><<<blocks, THREADS, SHARED_MEM, stream>>>( \
      A_ptr, B_ptr, C_ptr, s_ptr, \
      prob_m, prob_n, prob_k, \
      locks \
    ); \
  }

const int ERR_PROB_SHAPE = 1;
const int ERR_KERN_SHAPE = 2;

int marlin_cuda(
  const void* A,
  const void* B,
        void* C,
        void* s,
  int prob_m,
  int prob_n,
  int prob_k,
  void* workspace,
  int groupsize = -1,
  int dev = 0,
  hipStream_t stream = 0,
  int thread_k = -1,
  int thread_n = -1,
  int sms = -1,
  int max_par = 16
) {
  int tot_m = prob_m;
  int tot_m_blocks = ceildiv(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1)
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  if (thread_k == -1 || thread_n == -1) {
    if (prob_m <= 16) {
      thread_k = 128;
      thread_n = 128;
    } else {
      thread_k = 64;
      thread_n = 256;
    }
  }

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;
  int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
  int blocks = sms;

  if (prob_n % thread_n != 0 || prob_k % thread_k != 0 || (group_blocks != -1 && prob_k % group_blocks != 0))
    return ERR_PROB_SHAPE;
  if (prob_m == 0 || prob_n == 0 || prob_k == 0)
    return 0;

  const int4* A_ptr = (const int4*) A;
  const int4* B_ptr = (const int4*) B;
  int4* C_ptr = (int4*) C;
  const int4* s_ptr = (const int4*) s;

  int* locks = (int*) workspace;

  int ret = 0;
  for (int i = 0; i < tot_m_blocks; i += 4) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > 4) {
      par = (16 * thread_m_blocks - pad) / 64;
      if (par > max_par)
        par = max_par;
      prob_m = 64 * par;
      i += 4 * (par - 1);
      thread_m_blocks = 4;
    }

    if (false) {}
    CALL_IF(1,  8,  8, -1)
    CALL_IF(1,  8,  8,  8)
    CALL_IF(1, 16,  4, -1)
    CALL_IF(1, 16,  4,  8)
    CALL_IF(2, 16,  4, -1)
    CALL_IF(2, 16,  4,  8)
    CALL_IF(3, 16,  4, -1)
    CALL_IF(3, 16,  4,  8)
    CALL_IF(4, 16,  4, -1)
    CALL_IF(4, 16,  4,  8)
    else
      ret = ERR_KERN_SHAPE;

    A_ptr += 16 * thread_m_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
  }

  return ret;
}

bool load_gptq_file(const char* filepath, void** data, size_t* size) {
  std::ifstream file(filepath, std::ios::binary | std::ios::ate);
  if (!file.is_open()) return false;
  *size = file.tellg();
  file.seekg(0);
  *data = malloc(*size);
  file.read((char*)*data, *size);
  file.close();
  return true;
}

int main(int argc, char* argv[]) {
  // Use GPTQ dimensions
  const int M = 1;
  const int N = 11008;
  const int K = 2048;
  const int groupsize = (argc > 2) ? atoi(argv[2]) : -1;  // Allow groupsize as 2nd arg

  std::cout << "Testing with M=" << M << ", K=" << K << ", N=" << N << ", groupsize=" << groupsize << std::endl;

  // Allocate device memory
  void *d_A, *d_B, *d_C, *d_s, *d_workspace;
  size_t A_size = M * K * sizeof(half);
  size_t B_size = (K * N) / 2;  // 4-bit packed weights
  size_t C_size = M * N * sizeof(half);
  size_t s_size = (groupsize == -1) ? N * sizeof(half) : (K / groupsize) * N * sizeof(half);
  size_t workspace_size = (N / 128) * 16 * sizeof(int);

  hipMalloc(&d_A, A_size);
  hipMalloc(&d_B, B_size);
  hipMalloc(&d_C, C_size);
  hipMalloc(&d_s, s_size);
  hipMalloc(&d_workspace, workspace_size);

  // Initialize data
  std::vector<half> h_A(M * K);
  std::vector<int> h_B(B_size / sizeof(int));
  std::vector<half> h_s(s_size / sizeof(half));

  // TEST CONFIGURATION
  bool use_gptq_weights = (argc > 1 && argv[1][0] == 'w');
  bool use_gptq_scales = (argc > 1 && argv[1][0] == 's');
  bool use_both_gptq = (argc > 1 && argv[1][0] == 'b');

  std::cout << "\n=== Binary Search Test ===" << std::endl;
  if (use_both_gptq || use_gptq_weights) {
    std::cout << "Weights: GPTQ" << std::endl;
  } else {
    std::cout << "Weights: Random" << std::endl;
  }
  if (use_both_gptq || use_gptq_scales) {
    std::cout << "Scales: GPTQ" << std::endl;
  } else {
    std::cout << "Scales: Random" << std::endl;
  }
  std::cout << "==========================\n" << std::endl;

  // Input A - always use 0.1
  for (int i = 0; i < M * K; i++) {
    h_A[i] = __float2half(0.1f);
  }

  // Weights B
  if (use_both_gptq || use_gptq_weights) {
    void* gptq_data = nullptr;
    size_t gptq_size;
    if (load_gptq_file("up_proj_qweight.bin", &gptq_data, &gptq_size)) {
      // Unpack GPTQ and repack
      int* gptq_packed = (int*)gptq_data;
      std::vector<int> unpacked(K * N);
      for (int row = 0; row < 256; row++) {
        for (int col = 0; col < N; col++) {
          int packed_val = gptq_packed[row * N + col];
          for (int j = 0; j < 8; j++) {
            unpacked[(row * 8 + j) * N + col] = (packed_val >> (j * 4)) & 0xF;
          }
        }
      }
      for (size_t i = 0; i < h_B.size(); i++) {
        int val = 0;
        for (int j = 0; j < 8; j++) {
          val |= (unpacked[i * 8 + j] & 0xF) << (j * 4);
        }
        h_B[i] = val;
      }
      free(gptq_data);
      std::cout << "✓ Loaded GPTQ weights" << std::endl;
    } else {
      std::cerr << "Failed to load GPTQ weights, using random" << std::endl;
      use_gptq_weights = false;
    }
  }

  if (!use_gptq_weights && !use_both_gptq) {
    srand(42);
    for (size_t i = 0; i < h_B.size(); i++) {
      int val = 0;
      for (int j = 0; j < 8; j++) {
        val |= ((rand() % 16) & 0xF) << (j * 4);
      }
      h_B[i] = val;
    }
    std::cout << "✓ Generated random weights" << std::endl;
  }

  // Scales
  if (use_both_gptq || use_gptq_scales) {
    void* gptq_scales = nullptr;
    size_t scales_size;
    if (load_gptq_file("up_proj_scales.bin", &gptq_scales, &scales_size)) {
      memcpy(h_s.data(), gptq_scales, s_size);
      free(gptq_scales);
      std::cout << "✓ Loaded GPTQ scales" << std::endl;
    } else {
      std::cerr << "Failed to load GPTQ scales, using random" << std::endl;
      use_gptq_scales = false;
    }
  }

  if (!use_gptq_scales && !use_both_gptq) {
    srand(43);
    for (size_t i = 0; i < h_s.size(); i++) {
      h_s[i] = __float2half(0.01f + (float)rand() / RAND_MAX * 0.02f);
    }
    std::cout << "✓ Generated random scales" << std::endl;
  }

  hipMemcpy(d_A, h_A.data(), A_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B.data(), B_size, hipMemcpyHostToDevice);
  hipMemcpy(d_s, h_s.data(), s_size, hipMemcpyHostToDevice);
  hipMemset(d_workspace, 0, workspace_size);

  std::cout << "\nFirst 3 weights: " << h_B[0] << ", " << h_B[1] << ", " << h_B[2] << std::endl;
  std::cout << "First 3 scales: " << __half2float(h_s[0]) << ", " << __half2float(h_s[1]) << ", " << __half2float(h_s[2]) << std::endl;

  // Run the kernel
  std::cout << "\nRunning Marlin kernel..." << std::endl;
  int result = marlin_cuda(
    d_A, d_B, d_C, d_s,
    M, N, K,
    d_workspace,
    groupsize
  );

  if (result == 0) {
    printf("\n✓ Kernel executed successfully!\n");
    printf("Matrix multiplication: (%d x %d) * (%d x %d) = (%d x %d)\n", M, K, K, N, M, N);

    // Copy result back and check
    std::vector<half> h_C(M * N);
    hipMemcpy(h_C.data(), d_C, C_size, hipMemcpyDeviceToHost);

    printf("\nFirst 20 output values:\n");
    for (int i = 0; i < 20; i++) {
      printf("  C[%2d] = %.6f\n", i, __half2float(h_C[i]));
    }

    // Check if all zeros
    bool all_zero = true;
    float max_abs = 0;
    for (int i = 0; i < M * N; i++) {
      float val = std::abs(__half2float(h_C[i]));
      if (val > 1e-6) all_zero = false;
      if (val > max_abs) max_abs = val;
    }

    if (all_zero) {
      printf("\n⚠️  WARNING: All outputs are zero!\n");
    } else {
      printf("\n✓ SUCCESS: Found non-zero outputs! Max abs value: %.6f\n", max_abs);
    }

  } else if (result == ERR_PROB_SHAPE) {
    printf("Error: Problem shape incompatible with kernel constraints.\n");
  } else if (result == ERR_KERN_SHAPE) {
    printf("Error: No kernel implementation for these parameters.\n");
  }

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_s);
  hipFree(d_workspace);

  return result;
}